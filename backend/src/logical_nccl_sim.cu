#include "hip/hip_runtime.h"
#include "logical_nccl_sim.hpp"
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel
__global__ void allReduceKernel(const float* A, const float* B, float* C, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// Wrapper
namespace logical_nccl_sim {

void simulateAllReduce(float* A, float* B, float* C, size_t N, hipStream_t stream, int logical_id) {
    // Ensure correct device is set based on pointer
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, A);
    hipSetDevice(attr.device);

    // Debug: check first few values of A and B
    float hostA[5], hostB[5];
    hipMemcpy(hostA, A, 5 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostB, B, 5 * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "[SimNCCL Debug] Logical GPU " << logical_id << " pre-kernel A[0..4]: ";
    for (int i = 0; i < 5; ++i) std::cout << hostA[i] << " ";
    std::cout << "\n[SimNCCL Debug] Logical GPU " << logical_id << " pre-kernel B[0..4]: ";
    for (int i = 0; i < 5; ++i) std::cout << hostB[i] << " ";
    std::cout << std::endl;

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    allReduceKernel<<<blocks, threads, 0, stream>>>(A, B, C, N);
    hipStreamSynchronize(stream);

    float host_buf[5];
    hipMemcpy(host_buf, C, 5 * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "[SimNCCL] Logical GPU " << logical_id 
              << " first 5 elements after simulated AllReduce: ";
    for (int i = 0; i < 5; ++i) std::cout << host_buf[i] << " ";
    std::cout << std::endl;
}

} // namespace logical_nccl_sim

