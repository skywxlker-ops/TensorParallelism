#include "hip/hip_runtime.h"
// #include "task.hpp"
// #include <vector>
// #include <iostream>

// __global__ void initTensor(float* data, float val, int n) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < n) data[idx] = val;
// }

// void Task::runAllReduce(Mesh& mesh) {
//     int num_gpus = mesh.size();
//     int num_elements = 4;
//     size_t bytes = num_elements * sizeof(float);

//     std::vector<float*> d_data(num_gpus);
//     std::vector<float> h_output(num_elements);

//     // Allocate & initialize data
//     for (int i = 0; i < num_gpus; ++i) {
//         hipSetDevice(i);
//         hipMalloc(&d_data[i], bytes);
//         initTensor<<<1, num_elements>>>(d_data[i], float(i + 1), num_elements);
//     }

//     std::cout << "[Task] Performing AllReduce across " << num_gpus << " GPUs..." << std::endl;

//     // AllReduce: sum across GPUs
//     NCCL_CHECK(ncclGroupStart());
//     for (int i = 0; i < num_gpus; ++i) {
//         NCCL_CHECK(ncclAllReduce(
//             d_data[i], d_data[i],
//             num_elements, ncclFloat, ncclSum,
//             mesh.getComm(i), 0
//         ));
//     }
//     NCCL_CHECK(ncclGroupEnd());

//     // Copy result back to host and print
//     for (int i = 0; i < num_gpus; ++i) {
//         hipSetDevice(i);
//         hipMemcpy(h_output.data(), d_data[i], bytes, hipMemcpyDeviceToHost);
//         std::cout << "[GPU " << i << "] Output: ";
//         for (float v : h_output) std::cout << v << " ";
//         std::cout << std::endl;
//     }

//     for (int i = 0; i < num_gpus; ++i) hipFree(d_data[i]);
// }


#include "task.hpp"
#include <iostream>
#include <thread>

__global__ void initTensorKernel(float* data, float val, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride)
        data[i] = val;
}

void Task::initTensors(std::vector<float*>& d_data, Mesh& mesh, int num_elements) {
    int num_gpus = mesh.size();
    for (int i = 0; i < num_gpus; ++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(&d_data[i], num_elements * sizeof(float)));

        int threads = 256;
        int blocks = (num_elements + threads - 1) / threads;
        initTensorKernel<<<blocks, threads>>>(d_data[i], float(i + 1), num_elements);
        CUDA_CHECK(hipDeviceSynchronize());
    }
}

void Task::runAllReduce(Mesh& mesh, std::vector<float*>& d_data, int num_elements) {
    int num_gpus = mesh.size();
    std::cout << "[Task] Performing AllReduce across " << num_gpus << " GPUs..." << std::endl;

    std::vector<std::thread> threads(num_gpus);

    // launch all-reduces concurrently
    for (int i = 0; i < num_gpus; ++i) {
        threads[i] = std::thread([&, i]() {
            CUDA_CHECK(hipSetDevice(i));
            mesh.allReduce(d_data[i], num_elements);
        });
    }

    // wait for all threads to finish
    for (auto& t : threads) t.join();

    // print results
    std::vector<float> h_output(num_elements);
    for (int i = 0; i < num_gpus; ++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMemcpy(h_output.data(), d_data[i], num_elements * sizeof(float), hipMemcpyDeviceToHost));
        std::cout << "[GPU " << i << "] Output: ";
        for (int j = 0; j < 10; ++j) std::cout << h_output[j] << " ";
        std::cout << "..." << std::endl;
    }
}


